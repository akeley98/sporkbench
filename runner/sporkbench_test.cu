#include "hip/hip_runtime.h"
#include "sporkbench_test.hpp"

#include <bit>
#include <cassert>
#include <hipblas.h>
#include <stdexcept>
#include <stdint.h>
#include <stdio.h>
#include <vector>

namespace sporkbench {

namespace sporkbench_test {

enum class TestDataCode
{
    random = 0,
    batch_index_identity = 1,
    tiled_numbers = 2,
    signs_only = 3,
};

// Copied pseudo random number generation code.
// http://www.jcgt.org/published/0009/03/02/
// Hash Functions for GPU Rendering, Mark Jarzynski, Marc Olano, NVIDIA
__device__ uint64_t pcg3d(uint32_t x, uint32_t y, uint32_t z)
{
    x = x*1664525u + 1013904223u;
    y = y*1664525u + 1013904223u;
    z = z*1664525u + 1013904223u;

    x += y*z;
    y += z*x;
    z += x*y;

    x ^= x >> 16u;
    y ^= y >> 16u;
    z ^= z >> 16u;

    x += y*z;
    y += z*x;
    z += x*y;

    return x ^ uint64_t(y) << 12u ^ uint64_t(z) << 24u;
}

template <typename T>
__global__ void device_init_test_data(
        T* d_tensor, uint32_t batch_size, uint32_t major_extent, uint32_t minor_extent, TestDataCode code)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t tid_z = threadIdx.z + blockIdx.z * blockDim.z;
    for (uint32_t z = tid_z; z < batch_size; z += blockDim.z * gridDim.z) {
        for (uint32_t y = tid_y; y < major_extent; y += blockDim.y * gridDim.y) {
            for (uint32_t x = tid_x; x < minor_extent; x += blockDim.x * gridDim.x) {
                T value;
                switch (code) {
                  case TestDataCode::batch_index_identity:
                    value = x == y ? T(1 + z) : T(0);
                    break;
                  case TestDataCode::tiled_numbers:
                    value = T((x % 64) + 100 * (y % 64));
                    break;
                  case TestDataCode::signs_only:
                    {
                        // uniform choice between -1, 0, 1.
                        value = T(int(pcg3d(x, y, z + 20010106) % 3) - 1);
                    }
                    break;
                  case TestDataCode::random:
                  default:
                    {
                        const auto randbits = pcg3d(x, y, z + 20010106);
                        if (randbits % 100'000u == 0) {
                            // 1 in 100'000 chance of a "big" value (1000).
                            // This greatly reduces the chance that a genuine bug is mistaken for fp error.
                            value = T(1000);
                        }
                        else if (randbits % 4u != 0u) {
                            value = T(0);  // 75% chance of a 0
                        }
                        else {
                            // 25% chance of random value [0, 1], biased towards small numbers.
                            value = T((pcg3d(x, y, 19980724) % 1'000'000) * 1e-6f);
                            value = (value * value) * (value * value);
                        }
                    }
                    break;
                }
                d_tensor[z * major_extent * minor_extent + y * minor_extent + x] = value;
            }
        }
    }
}


template <typename T>
void launch_init_test_data(
        T* d_tensor, uint32_t batch_size, uint32_t major_extent, uint32_t minor_extent,
        TestDataCode code, hipStream_t stream)
{
    dim3 grid{(minor_extent + 255u) / 256u, major_extent, batch_size};
    dim3 block{256, 1, 1};
    device_init_test_data<<<grid, block, 0, stream>>>(d_tensor, batch_size, major_extent, minor_extent, code);
}


__global__ void device_compare_tensor_test_init_bitfield(unsigned long long* d_bitfield)
{
    *d_bitfield = UINT64_MAX;
}

// Requires that *d_bitfield is initialized to UINT64_MAX.
// Compare the two equal-sized matrices and, if any comparison failures, put the coordinates of the wrong value
// into *d_bitfield, packed as batch * M * N + n * M + m (linear_index).
template <typename TA, typename TB>
__global__ void device_compare_tensor_test(GemmSize size, const TA* d_a, const TB* d_b, bool exact,
                                           unsigned long long* d_bitfield)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    uint32_t tid_z = threadIdx.z + blockIdx.z * blockDim.z;
    for (uint32_t batch = tid_z; batch < size.L; batch += blockDim.z * gridDim.z) {
        for (uint32_t m = tid_y; m < size.M; m += blockDim.y * gridDim.y) {
            for (uint32_t n = tid_x; n < size.N; n += blockDim.x * gridDim.x) {
                const size_t linear_index = size_t(batch) * size.M * size.N + size_t(n) * size.M + m;
                bool correct;
                if (exact) {
                    correct = d_a[linear_index] == d_b[linear_index];
                }
                else {
                    float a = float(d_a[linear_index]);
                    float b = float(d_b[linear_index]);
                    correct = a * b >= 0.0f;  // Sign error, or inf/nan if wrong
                    if (correct) {
                        a = fabsf(a);
                        b = fabsf(b);
                        const float m = fminf(a, b);
                        const float M = fmaxf(a, b);
                        correct = M == 0 || M / m < (1.0f + 1/128.0f);
                    }
                }
                if (!correct) {
                    atomicMin(d_bitfield, (unsigned long long)linear_index);
                }
            }
        }
    }
}

template <typename TA, typename TB>
__device__ void print_tensor_neighborhood(GemmSize size, const TA* d_a, const TB* d_b,
                                          uint32_t batch, uint32_t m, uint32_t n)
{
    uint32_t m_min = m < 2 ? 0u : m - 2;
    uint32_t m_max = m + 2 >= size.M ? size.M - 1u : m + 2;
    uint32_t n_min = n < 2 ? 0u : n - 2;
    uint32_t n_max = n + 2 >= size.N ? n - 1u : n + 2;

    for (uint32_t cm = m_min; cm <= m_max; cm++) {
        for (uint32_t cn = n_min; cn <= n_max; cn++) {
            if (cn == n && cm == m) {
                printf("\x1b[1m");
            }
            const size_t linear_index = size_t(batch) * size.M * size.N + size_t(cn) * size.M + cm;
            printf("[%6g, %5g]  ", static_cast<float>(d_a[linear_index]), static_cast<float>(d_b[linear_index]));
            if (cn == n && cm == m) {
                printf("\x1b[0m");
            }
        }
        printf("\n");
    }
}



// Print info on wrong value from function above.
template <typename TA, typename TB>
__global__ void device_compare_tensor_test_print(GemmSize size, const TA* d_a, const TB* d_b,
                                                 unsigned long long* d_bitfield)
{
    unsigned long long linear_index = *d_bitfield;
    if (linear_index != UINT64_MAX) {
        const uint32_t batch = linear_index / (size.M * size.N);
        const uint32_t n = (linear_index / size.M) % size.N;
        const uint32_t m = linear_index % size.M;
        const float a = static_cast<float>(d_a[linear_index]);
        const float b = static_cast<float>(d_b[linear_index]);
        printf("\x1b[1m[batch=%u, m=%u, n=%u]\x1b[0m %g != %g (test != expected)\n", batch, m, n, double(a), double(b));

        print_tensor_neighborhood(size, d_a, d_b, batch, m, n);
    }
}

template <typename TA, typename TB>
bool launch_device_compare_tensor(
        GemmSize size, const char* proc_name, const TA* d_a, const TB* d_b, bool exact, hipStream_t stream)
{
    dim3 grid(unsigned(size.N + 127) / 128u, unsigned(size.M), unsigned(size.L));
    dim3 block(128, 1, 1);
    unsigned long long* d_bitfield = 0;
    hipMallocAsync(&d_bitfield, sizeof(*d_bitfield), stream);
    if (!d_bitfield) {
        fprintf(stderr, "Alloc of d_bitfield failed: %s:%i\n", __FILE__, __LINE__);
        return false;
    }
    device_compare_tensor_test_init_bitfield<<<1, 1, 0, stream>>>(d_bitfield);
    device_compare_tensor_test<<<grid, block, 0, stream>>>(size, d_a, d_b, exact, d_bitfield);
    unsigned long long h_bitfield;
    hipMemcpy(&h_bitfield, d_bitfield, sizeof(h_bitfield), hipMemcpyDeviceToHost);
    if (h_bitfield != UINT64_MAX) {
        printf("\x1b[31m\x1b[1mFAILED:\x1b[0m %s, L=%i, MNK=[%i, %i, %i], K_split=%i\n",
            proc_name, size.L, size.M, size.N, size.K_cluster * size.K_split, size.K_split);
        fflush(stdout);
        device_compare_tensor_test_print<<<1, 1, 0, stream>>>(size, d_a, d_b, d_bitfield);
    }
    hipFreeAsync(d_bitfield, stream);
    return h_bitfield == UINT64_MAX;
}


TestResult run_gemm_case_test_data(
        const GemmCase& gemm_case, hipblasHandle_t cublasH, GemmSize size,
        float* A, float* B, float* C_test, float* C_expected,
        TestDataCode A_code, TestDataCode B_code, int test_count, hipStream_t stream)
{
    const uint32_t L = uint32_t(size.L);
    const uint32_t M = uint32_t(size.M);
    const uint32_t N = uint32_t(size.N);
    const uint32_t K = uint32_t(size.K_split * size.K_cluster);
    launch_init_test_data(A, L, M, K, A_code, stream);
    launch_init_test_data(B, L, N, K, B_code, stream);
    run_cublas_gemm(cublasH, size, A, B, C_expected);

    std::vector<float> test_times(test_count);
    std::vector<hipEvent_t> test_events(test_count + 1);
    auto new_event = [stream]
    {
        hipEvent_t event{};
        if (const hipError_t err = hipEventCreate(&event)) {
            throw std::runtime_error("hipEventCreate failed\n");
        }
        hipEventRecord(event, stream);
        return event;
    };

    for (int test_i = 0; test_i < test_count; ++test_i) {
        if (test_i == 0) {
            test_events[0] = new_event();
        }
        gemm_case.run_function(cublasH, size, A, B, C_test);
        test_events[test_i + 1] = new_event();
    }
    const bool exact = A_code == TestDataCode::signs_only && B_code == TestDataCode::signs_only && K <= 4096;
    const bool passed = launch_device_compare_tensor(size, gemm_case.proc_name, C_test, C_expected, exact, stream);

    for (int test_i = 0; test_i < test_count; ++test_i) {
        hipEventElapsedTime(&test_times[test_i], test_events[test_i], test_events[test_i + 1]);
        hipEventDestroy(test_events[test_i]);
    }
    hipEventDestroy(test_events[test_count]);
    std::sort(&test_times[0], &test_times[test_count]);
    const double median_ms = test_times[test_count / 2];
    const double flops = double(L) * M * N * K * 2000.0 / median_ms;
    return TestResult{passed, flops};
}

}  // end namespace sporkbench_test

TestResult run_gemm_case(
        const GemmCase& gemm_case, hipblasHandle_t cublasH, GemmSize size,
        float* A, float* B, float* C_test, float* C_expected,
        bool warmup, int num_trials)
{
    using namespace ::sporkbench::sporkbench_test;
    const hipStream_t stream = 0;
    assert(num_trials > 0);

    // Fill output C matrices with garbage.
    hipMemsetAsync(C_test, 0xDD, sizeof(C_test[0]) * size.L * size.M * size.N);

    if (warmup) {
        run_gemm_case_test_data(
                gemm_case, cublasH, size, A, B, C_test, C_expected,
                TestDataCode::batch_index_identity, TestDataCode::tiled_numbers, 1, stream);
        run_gemm_case_test_data(
                gemm_case, cublasH, size, A, B, C_test, C_expected,
                TestDataCode::tiled_numbers, TestDataCode::batch_index_identity, 1, stream);
        run_gemm_case_test_data(
                gemm_case, cublasH, size, A, B, C_test, C_expected,
                TestDataCode::signs_only, TestDataCode::signs_only, 1, stream);
    }
    const TestResult result = run_gemm_case_test_data(
            gemm_case, cublasH, size, A, B, C_test, C_expected,
            TestDataCode::random, TestDataCode::random, num_trials, stream);
    hipStreamSynchronize(stream);
    hipError_t err = hipGetLastError();
    if (err) {
        throw std::runtime_error(hipGetErrorString(err));
    }
    return result;
}

}  // end namespace sporkbench
